// 2021 JLFR
// experiment with levenshtein distance on FASTA sequences on GPU
// result: not fast enough
// next: use more than one SM
// almost 75% slower on V100 wrt GTX1080!


#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstring>
#include <fstream>
#include <iostream>
#include <vector>

//==============================================================================
// pre: m >= n, len(buffer) >= 3 * n
//==============================================================================

__global__ void levenshteinKernel(int m, const uint8_t* const x, int n, const uint8_t* const y, int32_t* w, int32_t* z, int32_t* zp)
{
    // so, each thread should be responsible of ceil(n / numThreads) elements
    int numThreads = blockDim.x;
    int numSteps = n / numThreads + (n % numThreads != 0);

    int32_t* result = z;
    result[0] = 1.0f;

    for (int i = 0; i < m + n - 1; ++i) {

        int j0 = max(0, i - m + 1);
        int j1 = min(n - 1, i); // included, [j0, j1]

        for (int step = 0; step < numSteps; ++step) {
            int j = step * blockDim.x + threadIdx.x;
            if (j0 <= j && j <= j1) {
                int32_t v = (0 == j || i == j ? i : zp[j - 1]) + (x[i - j] == y[j] ? 0 : 1);
                v = 0 == j ? v : min(v, 1 + z[j - 1]);
                v = i == j ? v : min(v, 1 + z[j]);
                w[j] = v;
            }
        }
        int32_t* tmp = zp;
        zp = z;
        z = w;
        w = tmp;
        __syncthreads();
    }

    if (0 == threadIdx.x) {
        result[0] = z[n - 1];
    }
}

/*
const std::string readFasta(const char* const path)
{
    std::ifstream stream(path, std::ios_base::in | std::ios_base::binary);
    std::string sequence;
    std::getline(stream, sequence);
    std::getline(stream, sequence);
    return sequence;
}
*/

void runExperiment(const uint8_t* pX, const uint8_t* pY)
{
    int m = strlen(reinterpret_cast<const char*>(pX));
    int n = strlen(reinterpret_cast<const char*>(pY));

    if (m < n) {
        std::swap(m, n);
        std::swap(pX, pY);
    }

    size_t xPitch = 0;
    uint8_t* x = nullptr;
    hipMallocPitch(&x, &xPitch, m, 2);

    uint8_t* y = x + xPitch;
    hipMemcpy(x, pX, m * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(y, pY, n * sizeof(uint8_t), hipMemcpyHostToDevice);

    size_t bufferPitch = 0;
    int32_t* buffer = nullptr;
    hipMallocPitch(&buffer, &bufferPitch, n * sizeof(int32_t), 3);

    int32_t* const w = buffer;
    int32_t* const z = reinterpret_cast<int32_t*>(reinterpret_cast<uint8_t*>(w) + bufferPitch);
    int32_t* const zp = reinterpret_cast<int32_t*>(reinterpret_cast<uint8_t*>(z) + bufferPitch);

    levenshteinKernel<<<1, 1024>>>(m, x, n, y, w, z, zp);

    hipDeviceSynchronize();

    int32_t result = 0;
    hipMemcpy(&result, z, sizeof(int32_t), hipMemcpyDeviceToHost);

    hipFree(buffer);
    hipFree(x);

    std::cout << "Result: " << result << std::endl;
}

int main(int argc, char* argv[])
{
    assert(3 == argc);
    const auto sequence1 = reinterpret_cast<const uint8_t*>(argv[1]);
    const auto sequence2 = reinterpret_cast<const uint8_t*>(argv[2]);

    runExperiment(sequence1, sequence2);

    return EXIT_SUCCESS;
}
